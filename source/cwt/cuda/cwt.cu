#include "hip/hip_runtime.h"
/// \file cwt.cpp
/// \author Denis Kozlov
#include <chrono>
#include <hipfft/hipfft.h>
#include <stdio.h>
#include <iostream>
#include <cwt/cuda/cwt.h>


#define BLOCK_DIM 128
namespace Kernels
{
    __global__ void conv(float *out, const float *in, const float *filter, int cols, int rows, int fcols, int frows)
    {
        int idx_x = blockDim.x * blockIdx.x + threadIdx.x;
        int idx_y = blockDim.y * blockIdx.y + threadIdx.y;

        int pad_rows = frows / 2;
        int pad_cols = fcols / 2;
        int tile_rows = BLOCK_DIM + 2 * pad_rows;
        int tile_cols = BLOCK_DIM + 2 * pad_cols;

        extern __shared__ float s_input[];

        for (int row = 0; row <= tile_rows / BLOCK_DIM; row++)
        {
            for (int col = 0; col <= tile_cols / BLOCK_DIM; col++)
            {
                int idx_row = idx_y + BLOCK_DIM * row - pad_rows;
                int idx_col = idx_x + BLOCK_DIM * col - pad_cols;
                int fid_row = threadIdx.y + BLOCK_DIM * row;
                int fid_col = threadIdx.x + BLOCK_DIM * col;

                if (fid_row >= tile_rows || fid_col >= tile_cols)
                {
                    continue;
                }

                s_input[tile_cols * fid_row + fid_col] = \
                        (idx_row >= 0 && idx_row < rows && idx_col >= 0 && idx_col < cols) ?
                        in[cols * idx_row + idx_col] : 0.f;
            }
        }

        __syncthreads();

        if (idx_x < cols)
        {
            if (idx_y < rows)
            {
                float result = 0.f;
                for (int filter_row = -frows / 2; filter_row <= frows / 2; ++filter_row)
                {
                    for (int filter_col = -fcols / 2; filter_col <= fcols / 2; ++filter_col)
                    {
                        int image_row = threadIdx.y + pad_rows + filter_row;
                        int image_col = threadIdx.x + pad_cols + filter_col;

                        float image_value = s_input[tile_cols * image_row + image_col];
                        float filter_value = filter[(filter_row + frows / 2) * fcols + filter_col + fcols / 2];

                        result += image_value * filter_value;
                    }
                }

                int index = idx_y * cols + idx_x;
                out[index] = result;
            }
        }
    }

    __global__
    void rearrange(float* input, float* output, int cols, int rows, int frows)
    {
        int i = blockDim.x * blockIdx.x + threadIdx.x;

        int col  = i % cols;
        int row  = (i / cols) % rows;
        int frow = i / (cols * rows);

        if (frow < frows)
        {
            int in_idx  = col + cols * row + cols * rows * frow;
            int out_idx = col + cols * frow + cols * frows * row;
            output[out_idx] = input[in_idx];

        }
    }

    __inline__ __device__
    hipfftComplex operator*(hipfftComplex const& a, hipfftComplex const& b) {
        hipfftComplex c;
        c.x = a.x * b.x - a.y * b.y;
        c.y = a.x * b.y + a.y * b.x;
        return c;
    }

    __global__ void multiply(hipfftComplex* in, hipfftComplex* wavelet, hipfftComplex* out, int len)
    {
        int i = blockDim.x * blockIdx.x + threadIdx.x;

        if (i < len)
        {
            out[i] = in[i] * wavelet[i];
        }
    }

    void createStreams(hipStream_t *stream, int count)
    {
        for (int i = 0; i < count; i++)
        {
            hipStreamCreate(&stream[i]);
        }
    }
}

CUDA::CUDA(int cols, int rows, const Wavelet& wavelet, const std::vector<float>& scales)
    : CWT(cols, rows, wavelet, scales)
{

}

CUDA::~CUDA()
{

}

bool execute_conv(const float* in, float* out, int cols, int rows, const Wavelet& wavelet)
{
    auto err = hipSetDevice(0);
    if (err != 0) std::cout << err << std::endl;
    int fcols = wavelet.shape(0);
    int frows = wavelet.shape(1);

    float *d_in;
    float *d_out;
    float *d_out_rearranged;
    float *d_wavelet;


    err = hipMalloc(&d_in, cols * rows * sizeof(float));
        if (err != 0) std::cout << err << std::endl;

    err = hipMalloc(&d_out, cols * rows * frows * sizeof(float));
        if (err != 0) std::cout << err << std::endl;

    err = hipMalloc(&d_out_rearranged, cols * rows * frows * sizeof(float));
    if (err != 0) std::cout << err << std::endl;

    err = hipMalloc(&d_wavelet, fcols * frows * sizeof(float));
        if (err != 0) std::cout << err << std::endl;

    err = hipMemcpy(d_in, in, cols * rows * sizeof(float), hipMemcpyHostToDevice);
        if (err != 0) std::cout << err << std::endl;

    err = hipMemcpy(d_wavelet, wavelet.ptr(), fcols * frows * sizeof(float), hipMemcpyHostToDevice);
        if (err != 0) std::cout << err << std::endl;


    hipStream_t stream[frows];
    Kernels::createStreams(stream, frows);

    auto start = std::chrono::high_resolution_clock::now();

    dim3 block(BLOCK_DIM, 1);
    dim3 grid(std::ceil((float) (cols + BLOCK_DIM - 1) / BLOCK_DIM), rows);

    int shared_mem = (2 * fcols + BLOCK_DIM) * sizeof(float);

    for (int i = 0; i < frows; i++)
    {
        // Shape frows x rows x cols !
        Kernels::conv<<<grid, block, shared_mem>>>
            (d_out + i * cols * rows, d_in, d_wavelet + i * wavelet.shape(0), cols, rows, wavelet.shape(0), 1);
    }


    int blocks = ceil((float) cols * rows * frows / 128.0f);
    Kernels::rearrange<<<blocks, 128>>>(d_out, d_out_rearranged, cols, rows, frows);

    hipDeviceSynchronize();
    auto finish = std::chrono::high_resolution_clock::now();
    auto duration_us = std::chrono::duration_cast<std::chrono::microseconds>(finish - start);
    auto duration_ms = std::chrono::duration_cast<std::chrono::milliseconds>(finish - start);
    std::cout << "CUDA Conv Elapsed time: " << duration_us.count() << "us\t" << duration_ms.count() << "ms\n";

    err = hipMemcpy(out, d_out_rearranged, cols * rows * frows * sizeof(float), hipMemcpyDeviceToHost);
            if (err != 0) std::cout << err << std::endl;

    hipFree(d_in);
    hipFree(d_out);
    hipFree(d_wavelet);
    return true;
}
//
bool execute_fft(const float* in, float* out, int cols, int rows, const Wavelet& wavelet)
{
    auto err = hipSetDevice(0);
    if (err != 0) std::cout << err << std::endl;
    int fcols = wavelet.shape(0);
    int frows = wavelet.shape(1);

    float *d_in;
    hipfftComplex *fft_in;

    float *d_out;
    hipfftComplex *fft_out;

    float *d_wavelet;
    hipfftComplex *fft_wavelet;

    int M = (floor(cols / 2) + 1);
    err = hipMalloc(&d_in, cols * rows * sizeof(float));
    if (err != 0) std::cout << err << std::endl;

    err = hipMalloc(&fft_in, M * rows * sizeof(hipfftComplex));
    if (err != 0) std::cout << err << std::endl;

    err = hipMalloc(&d_out, cols * rows * frows * sizeof(float));
    if (err != 0) std::cout << err << std::endl;

    err = hipMalloc(&fft_out, M * rows * frows * sizeof(hipfftComplex));
    if (err != 0) std::cout << err << std::endl;

    err = hipMalloc(&d_wavelet, cols * frows * sizeof(float));
    if (err != 0) std::cout << err << std::endl;

    err = hipMalloc(&fft_wavelet, M * frows * sizeof(hipfftComplex));
    if (err != 0) std::cout << err << std::endl;


    err = hipMemcpy(d_in, in, cols * rows * sizeof(float), hipMemcpyHostToDevice);
    if (err != 0) std::cout << err << std::endl;

    hipMemset(d_wavelet, 0.0f, cols * frows);
    for (int i = 0; i < frows; i++)
    {
        err = hipMemcpy(d_wavelet + i * cols, wavelet.ptr() + i * fcols, fcols * sizeof(float), hipMemcpyHostToDevice);
        if (err != 0) std::cout << err << std::endl;
    }


//    hipStream_t stream[scales.size()];
//    Kernels::createStreams(stream, scales.size());

    hipfftHandle signal_plan;
    auto cerr = hipfftPlan1d(&signal_plan, cols, HIPFFT_R2C, rows);
    if (cerr != 0) std::cout << cerr << std::endl;

    hipfftHandle wavelet_plan;
    cerr = hipfftPlan1d(&wavelet_plan, cols, HIPFFT_R2C, frows);
    if (cerr != 0) std::cout << cerr << std::endl;

    hipfftHandle out_plan;
    cerr = hipfftPlan1d(&out_plan, cols, HIPFFT_C2R, rows * frows);
    if (cerr != 0) std::cout << cerr << std::endl;


    auto start = std::chrono::high_resolution_clock::now();

    cerr = hipfftExecR2C(signal_plan, d_in, fft_in);
    if (cerr != 0) std::cout << cerr << std::endl;

    cerr = hipfftExecR2C(wavelet_plan, d_wavelet, fft_wavelet);
    if (cerr != 0) std::cout << cerr << std::endl;

    for (int c = 0; c < rows; c++)
    {
        for (int a = 0; a < frows; a++)
        {
            Kernels::multiply<<<M, 128>>>(fft_in + c * M, fft_wavelet + a * M, fft_out + a * M + c * M * frows, M);
        }
    }

    cerr = hipfftExecC2R(out_plan, fft_out, d_out);
    if (cerr != 0) std::cout << cerr << std::endl;

    hipDeviceSynchronize();
    auto finish = std::chrono::high_resolution_clock::now();
    auto duration_us = std::chrono::duration_cast<std::chrono::microseconds>(finish - start);
    auto duration_ms = std::chrono::duration_cast<std::chrono::milliseconds>(finish - start);
    std::cout << "CUDA FFT Elapsed time: " << duration_us.count() << "us\t" << duration_ms.count() << "ms\n";

    err = hipMemcpy(out, d_out, cols * rows * frows * sizeof(float), hipMemcpyDeviceToHost);
    if (err != 0) std::cout << err << std::endl;

    hipfftDestroy(wavelet_plan);
    hipfftDestroy(signal_plan);
    hipfftDestroy(out_plan);
    hipFree(d_in);
    hipFree(d_out);
    hipFree(d_wavelet);
    hipFree(fft_in);
    hipFree(fft_out);
    hipFree(fft_wavelet);
    return true;
}


bool CUDA::execute(const float* in, float* out, Mode mode)
{
    bool status = false;
    if (mode == TimeDomain)
    {
        status = execute_conv(in, out, cols, rows, wavelet);
    }
    else if (mode == FrequencyDomain)
    {
        status = execute_fft(in, out, cols, rows, wavelet);
    }

    return status;
}